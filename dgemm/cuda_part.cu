#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <assert.h>
#include <mkl.h>

#include "headers.h"


extern "C" void CUBLAS_DGEMM_(const CBLAS_LAYOUT 	 layout,
                   const CBLAS_TRANSPOSE TRANSA,
                   const CBLAS_TRANSPOSE TRANSB,
                   const int 	              m,
                   const int 	              n,
                   const int 	              k,
                   const double           alpha,
                   const double              *A,
                   const int 	            lda,
                   const double              *B,
                   const int 	            ldb,
                   const double            beta,
                   double                    *C,
                   const int 	            ldc)
{
	hipblasHandle_t handle;
	hipblasOperation_t transa, transb;
	hipblasStatus_t cubSt;
	hipError_t cuE;

	double *d_A, *d_B, *d_C;
	switch (TRANSA)
	{
		case CblasNoTrans:   transa = HIPBLAS_OP_N; break;
		case CblasTrans:     transa = HIPBLAS_OP_T; break;
		case CblasConjTrans: transa = HIPBLAS_OP_C; break;
		default: printf("cublas_dgemm: Fatal Error. TRANSA not recognized.\n"); exit(0);
	}
	
	switch (TRANSB)
	{
		case CblasNoTrans:   transb = HIPBLAS_OP_N; break;
		case CblasTrans:     transb = HIPBLAS_OP_T; break;
		case CblasConjTrans: transb = HIPBLAS_OP_C; break;
		default: printf("cublas_dgemm: Fatal Error. TRANSB not recognized.\n"); exit(0);
	}

	while ((cuE = hipMalloc((void **) &d_A, sizeof(double) * m * k)) != hipSuccess) continue; assert(hipSuccess == cuE);
	while ((cuE = hipMalloc((void **) &d_B, sizeof(double) * k * n)) != hipSuccess) continue; assert(hipSuccess == cuE);
	while ((cuE = hipMalloc((void **) &d_C, sizeof(double) * m * n)) != hipSuccess) continue; assert(hipSuccess == cuE);

	while ((cubSt = hipblasCreate(&handle)) != HIPBLAS_STATUS_SUCCESS) continue; assert(HIPBLAS_STATUS_SUCCESS == cubSt);
	while ((cubSt = hipblasSetMatrix(m, k, sizeof(*A), A, m, d_A, m)) != HIPBLAS_STATUS_SUCCESS) continue; assert(HIPBLAS_STATUS_SUCCESS == cubSt);
	while ((cubSt = hipblasSetMatrix(k, n, sizeof(*B), B, k, d_B, k)) != HIPBLAS_STATUS_SUCCESS) continue; assert(HIPBLAS_STATUS_SUCCESS == cubSt);
	
	while ((cubSt = hipblasDgemm(handle, transa, transb, m, n, k, &alpha, d_A, lda, d_B, ldb, &beta, d_C, ldc)) != HIPBLAS_STATUS_SUCCESS) continue;
	assert(HIPBLAS_STATUS_SUCCESS == cubSt);

	while ((cubSt = hipblasGetMatrix(m, n, sizeof(*C), d_C, m, C, m)) != HIPBLAS_STATUS_SUCCESS) continue; assert(HIPBLAS_STATUS_SUCCESS == cubSt);

	while ((cuE = hipFree(d_A)) != hipSuccess) continue; assert(hipSuccess == cuE);
	while ((cuE = hipFree(d_B)) != hipSuccess) continue; assert(hipSuccess == cuE);
	while ((cuE = hipFree(d_C)) != hipSuccess) continue; assert(hipSuccess == cuE);
	
	while ((cubSt = hipblasDestroy(handle)) != HIPBLAS_STATUS_SUCCESS) continue; assert(HIPBLAS_STATUS_SUCCESS == cubSt);
 	while ((cuE = hipDeviceSynchronize()) != hipSuccess) continue; assert(hipSuccess == cuE);
}